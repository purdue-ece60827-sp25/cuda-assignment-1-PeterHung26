#include "hip/hip_runtime.h"

#include "cudaLib.cuh"
#include "cpuLib.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort)
{
	if (code != hipSuccess) 
	{
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

__global__ 
void saxpy_gpu (float* x, float* y, float scale, int size) {
	//	Insert GPU SAXPY kernel code here
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if(idx < size)
		y[idx] = scale * x[idx] + y[idx];
}

int runGpuSaxpy(int vectorSize) {

	std::cout << "Hello GPU Saxpy!\n";

	//	Insert code here
	std::cout << "Lazy, you are!\n";
	std::cout << "Write code, you must\n";
	// device_prop();
	int vectorByte = vectorSize * sizeof(float);
	// Set up the thread block
	dim3 DimGrid(ceil(vectorSize/256.0),1,1);
	dim3 DimBlock(256, 1, 1);
	// Memory Allocation
	// Variable on host (for the verification)
	float * x, * y, * veri,  scale;
	x = (float *) malloc(vectorByte);
	y = (float *) malloc(vectorByte);
	veri = (float *) malloc(vectorByte);
	float * x_d, * y_d;
	hipMalloc((void **) &x_d, vectorByte);
	hipMalloc((void **) &y_d, vectorByte);
	scale = 2.0f;
	vectorInit(x, vectorSize);
	vectorInit(y, vectorSize);
	hipMemcpy(x_d, x, vectorByte, hipMemcpyHostToDevice);
	hipMemcpy(y_d, y, vectorByte, hipMemcpyHostToDevice);
	// Kernel invocation code
	saxpy_gpu<<<DimGrid, DimBlock>>>(x_d, y_d, scale, vectorSize);
	// Check the result with CPU
	hipMemcpy(veri, y_d, vectorByte, hipMemcpyDeviceToHost);
	int errorCount = verifyVector(x, y, veri, scale, vectorSize);
	std::cout << "Found " << errorCount << " / " << vectorSize << " errors \n";
	// Free the host and device memory
	free(x);
	free(y);
	free(veri);
	hipFree(x_d);
	hipFree(y_d);
	return 0;
}

/* 
 Some helpful definitions

 generateThreadCount is the number of threads spawned initially. Each thread is responsible for sampleSize points. 
 *pSums is a pointer to an array that holds the number of 'hit' points for each thread. The length of this array is pSumSize.

 reduceThreadCount is the number of threads used to reduce the partial sums.
 *totals is a pointer to an array that holds reduced values.
 reduceSize is the number of partial sums that each reduceThreadCount reduces.

*/

__global__
void generatePoints (uint64_t * pSums, uint64_t pSumSize, uint64_t sampleSize) {
	//	Insert code here
}

__global__ 
void reduceCounts (uint64_t * pSums, uint64_t * totals, uint64_t pSumSize, uint64_t reduceSize) {
	//	Insert code here
}

int runGpuMCPi (uint64_t generateThreadCount, uint64_t sampleSize, 
	uint64_t reduceThreadCount, uint64_t reduceSize) {

	//  Check CUDA device presence
	int numDev;
	hipGetDeviceCount(&numDev);
	if (numDev < 1) {
		std::cout << "CUDA device missing!\n";
		return -1;
	}

	auto tStart = std::chrono::high_resolution_clock::now();
		
	float approxPi = estimatePi(generateThreadCount, sampleSize, 
		reduceThreadCount, reduceSize);
	
	std::cout << "Estimated Pi = " << approxPi << "\n";

	auto tEnd= std::chrono::high_resolution_clock::now();

	std::chrono::duration<double> time_span = (tEnd- tStart);
	std::cout << "It took " << time_span.count() << " seconds.";

	return 0;
}

double estimatePi(uint64_t generateThreadCount, uint64_t sampleSize, 
	uint64_t reduceThreadCount, uint64_t reduceSize) {
	
	double approxPi = 0;

	//      Insert code here
	std::cout << "Sneaky, you are ...\n";
	std::cout << "Compute pi, you must!\n";
	return approxPi;
}

// Remeber to hide it before submission
int device_prop(){
	hipDeviceProp_t prop;
    int device;
    hipError_t err = hipGetDevice(&device);
	if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
        return -1;
    }
    hipGetDeviceProperties(&prop, device);

	int maxActiveBlocks;
    int blockSize = 128;  // Example block size (you can vary this)
    size_t sharedMemoryPerBlock = 0; // Default shared memory usage

    // Get maximum active blocks per SM
    hipOccupancyMaxActiveBlocksPerMultiprocessor(
        &maxActiveBlocks, saxpy_gpu, blockSize, sharedMemoryPerBlock
    );

    int totalThreadsPerSM = maxActiveBlocks * blockSize;
    float occupancy = (float)totalThreadsPerSM / prop.maxThreadsPerMultiProcessor * 100.0f;

    printf("Device Name: %s\n", prop.name);
    printf("Number of SMs: %d\n", prop.multiProcessorCount);
    printf("Max Active Blocks per SM: %d\n", maxActiveBlocks);
    printf("Total Threads per SM: %d\n", totalThreadsPerSM);
    printf("Max Threads per SM: %d\n", prop.maxThreadsPerMultiProcessor);
    printf("Occupancy: %.2f%%\n", occupancy);

    return 0;
}